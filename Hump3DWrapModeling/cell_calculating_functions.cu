#include "hip/hip_runtime.h"
#include "SimulationParams.h"
#include "3D_stencil.cuh"

__device__ __host__ int indexof(int x, int y, int z, SimulationParams *params) {
    return x * (params->dims[0] * params->dims[1]) + y * params->dims[1] + z;
}

__device__ __host__ int3 indexof(int id, SimulationParams *params) {
    int3 res;
    res.x = id / (params->dims[0] * params->dims[1]);
    res.z = id % params->dims[1];
    res.y = (id - res.x * (params->dims[0] * params->dims[1]) - res.z) / params->dims[1];
    return res;
}

__device__ double relaxed_derivative(double a, double derivative_left, double derivative_right) {
    return derivative_left * (a + abs(a)) / 2 +
           derivative_right * (a - abs(a)) / 2;
}

__device__ double mu(double xi1, double xi2, SimulationParams *params) {
    return params->A * exp(-pow(xi1, 2.0) * params->alpha
                           - pow(xi2, 2.0) * params->beta);
}

__device__ double mu_derivative(double xi1, double xi2, int dim, SimulationParams *params) {
    if (dim == 0)
        return -2 * params.A * xi1 * params.alpha * exp(-pow(xi1, 2.0) * params.alpha
                                                        - pow(xi2, 2.0) * params.beta);
    else
        return -2 * params.A * xi2 * params.beta * exp(-pow(xi1, 2.0) * params.alpha
                                                       - pow(xi2, 2.0) * params.beta);
}

__device__ double
u_point(double h_point, double c, double theta, double xi1, double xi2, SimulationParams *params) {
    return h_point + c * (theta + mu(xi1, xi2, params));
}

__device__ double
H_point(double u_point, double c, double theta, double xi1, double xi2, SimulationParams *params) {
    return u_point - c * (theta + mu(xi1, xi2, params));
}

__device__ double left_derivative(double left, double right, double delta) {
    return (right - left) / delta;
}

__device__ double
H_point(Stencil3D *__restrict__ H, Stencil3D *__restrict__ W, Stencil3D *__restrict__ V, SimulationParams *params) {
    // H->center.y = theta
    // H->center.x = xi1
    // H->center.z = xi2
    return H->center.w
           - params->timeStep * (
            relaxed_derivative(
                    H->center.w + c * (H->center.y + mu(H->center.x, H->center.z, params)),
                    H->dx_l(),
                    H->dx_r())
            + relaxed_derivative(
                    (H->center.w + c * (H->center.y + mu(H->center.x, H->center.z, params))) *
                    mu_derivative(H->center.x, H->center.z, 0, params) +
                    W->center.w * mu_derivative(H->center.x, H->center.z, 1, params) - V->center.w,
                    H->dy_l(),
                    H->dy_r())
            + relaxed_derivative(
                    W->center.w,
                    H->dz_l(),
                    H->dz_r())
            + V->y_less.w * c + V->center.w * c -
            H->dy2()
    );
}

__device__ double
W_point(Stencil3D *__restrict__ H, Stencil3D *__restrict__ W, Stencil3D *__restrict__ V, SimulationParams *params) {
    return W->center.w
           - params->timeStep * (
            relaxed_derivative(
                    W->center.w,
                    W->dz_l(),
                    W->dz_r())
            + relaxed_derivative(
                    H->center.w + c * (H->center.y + mu(H->center.x, H->center.z, params)),
                    W->dx_l(),
                    W->dx_r())
            + relaxed_derivative(
                    V->center.w + W->center.w * mu_derivative(H->center.x, H->center.z, 1, params) +
                    mu_derivative(H->center.x, H->center.z, 0, params) *
                    (H->center.w + c * (H->center.y + mu(H->center.x, H->center.z, params))),
                    W->dy_l(),
                    W->dy_r())
            - W->dy2());
}

