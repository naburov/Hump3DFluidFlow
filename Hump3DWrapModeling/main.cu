#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS

void process_one_config(const char *cnf_path);

void process_one_config_cuda(const char *cnf_path);

#include <iomanip>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <fstream>

#include "export_functions.h"
#include "Config.h"
#include "SimulationParams.h"

#include <vector>
#include <algorithm>
#include<stdio.h>
#include<stdlib.h>

#ifdef __APPLE__

#include <sys/stat.h>

#elif _WIN32
#include "direct.h"
#elif __linux__

#include <sys/stat.h>

#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include "test_kernels.cuh"
#include "cell_calculating_functions.cuh"
#include "calculating_kernels.cuh"

// TODO: graph execution
// TODO: time becnchmarks - done
// TODO: dp/dxi
// TODO: +/- in H_kernel
// TODO: reduce_max_kernel

void filecopy(FILE *dest, FILE *src) {
    const int size = 16384;
    char      buffer[size];

    while (!feof(src)) {
        int n = fread(buffer, 1, size, src);
        fwrite(buffer, 1, n, dest);
    }

    fflush(dest);
}

int main(int argc, char *argv[]) {

    std::vector<std::string> cnfs;
    for (int                 i = 1; i < argc; ++i) {
        cnfs.emplace_back(argv[i]);
    }

    // /tmp/tmp.SJNETVCrGC/cmake-build-release-x86_64-llvm-homewsl/Hump3DFluidFlow
    // /tmp/tmp.SJNETVCrGC/configs/testconf

    for (const auto &cnf: cnfs) {
        std::cout << "Processing cnf " << cnf << std::endl;
        process_one_config_cuda(cnf.c_str());
    }
    std::cout << "Done processing";
    return 0;
}

void process_one_config_cuda(const char *cnf_path) {
    //region ConfigParsing
    auto cnf = Config(cnf_path);
    cnf.print();

    auto      t_dims = cnf.get_grid_dimensions();
    const int dims[] = {t_dims[0], t_dims[1], t_dims[2]};

    auto t_sizes = cnf.get_grid_sizes();


    const double time_step = cnf.get_timestep();

    auto func_params = cnf.get_function_params();

    SimulationParams sim_params = {
            {t_dims[0], t_dims[1], t_dims[2]},
            {t_sizes[1], t_sizes[5], t_sizes[3]},
            {
                    (t_sizes[0] - t_sizes[1]) / (t_dims[0]),
                    (t_sizes[4] - t_sizes[5]) / (t_dims[1]),
                    (t_sizes[2] - t_sizes[3]) / (t_dims[2])
            },
            func_params[0],
            func_params[1],
            cnf.get_hump_height(),
            time_step
    };

    auto t_params    = cnf.get_saving_params();
    auto save_every  = t_params[0];
    auto print_every = t_params[1];
    auto max_steps   = cnf.get_max_steps();
    //endregion ConfigParsing

    printf("Config is parsed\n");

    //region FileManagementInit
    auto t  = time(nullptr);
    auto tm = *localtime(&t);

    std::ostringstream oss;
    oss << std::put_time(&tm, "%d-%m-%Y %H-%M-%S");
    auto str = oss.str();

    std::stringstream ss;
    const std::string filename = "./out " + str + "/output";

#ifdef __APPLE__
    mkdir(("./out " + str).c_str(), 0777);
#elif _WIN32
    _mkdir(("./out " + str).c_str());
#elif __linux__
    mkdir(("./out " + str).c_str(), 0777);
#endif

    ss.str(std::string());
    ss << filename << "-cnf";

    FILE *infile  = fopen(cnf_path, "rb");
    FILE *outfile = fopen(ss.str().c_str(), "wb");

    filecopy(outfile, infile);

    fclose(infile);
    fclose(outfile);

    ss.str(std::string());
    ss << filename << "_grid.vts";
    export_grid(ss.str(), sim_params);
    //endregion FileManagementInit

    printf("Folders are created\n");

    //region cpu tensors allocation
    size_t grid_size_bytes = t_dims[0] * t_dims[1] * t_dims[2] * sizeof(double);
    auto   U               = (double *) malloc(grid_size_bytes);
    auto   H               = (double *) malloc(grid_size_bytes);
    auto   W               = (double *) malloc(grid_size_bytes);
    auto   V               = (double *) malloc(grid_size_bytes);
    //endregion U and W initialization

    //region CudaMemoryInitTransfer
    auto             num_threads_per_block = 512;
    auto             num_blocks            = static_cast<int>(std::ceil(
            static_cast<double>(t_dims[0] * t_dims[1] * t_dims[2]) / num_threads_per_block));
    double           *d_U, *d_H, *d_W, *d_V,
                     *d_old_U, *d_old_H, *d_old_W, *d_old_V;
    SimulationParams *d_sim_params;
    hipMalloc(&d_H, grid_size_bytes);
    hipMalloc(&d_U, grid_size_bytes);
    hipMalloc(&d_W, grid_size_bytes);
    hipMalloc(&d_V, grid_size_bytes);

    hipMalloc(&d_old_H, grid_size_bytes);
    hipMalloc(&d_old_U, grid_size_bytes);
    hipMalloc(&d_old_W, grid_size_bytes);
    hipMalloc(&d_old_V, grid_size_bytes);

    hipMalloc(&d_sim_params, sizeof(SimulationParams));

    hipMemcpy(d_sim_params, &sim_params, sizeof(SimulationParams), hipMemcpyHostToDevice);
    hipMemcpy(d_H, H, grid_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, grid_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, grid_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, grid_size_bytes, hipMemcpyHostToDevice);

    auto err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stdout, "Error occured while memorytransfer, %s \n", hipGetErrorString(err));
    }
    //endregion CudaMemoryTransfer

    //region initial conditions
    u_init_kernel<<<num_blocks, num_threads_per_block>>>(d_U, d_sim_params);
    err = hipGetLastError();

    w_init_kernel<<<num_blocks, num_threads_per_block>>>(d_W, d_sim_params);
    err = hipGetLastError();

    h_kernel<<<num_blocks, num_threads_per_block>>>(d_U, d_H, d_sim_params);
    err = hipGetLastError();

    v_func_kernel<<<num_blocks, num_threads_per_block>>>(d_W, d_V, d_U, d_sim_params);
    err = hipGetLastError();

    integrate_v_kernel<<<num_blocks, num_threads_per_block>>>(d_V, d_sim_params);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stdout, "Error occured, %s \n", hipGetErrorString(err));
    }
    //endregion initial conditions

    hipDeviceSynchronize();

    auto stop     = false;
    auto it_count = 0;

    hipEvent_t kernel_event_start, kernel_event_stop, it_start, it_stop;

    std::vector<float> h_times;
    std::vector<float> u_times;
    std::vector<float> w_times;
    std::vector<float> v_times;
    std::vector<float> it_times;

    hipEventCreate(&kernel_event_start);
    hipEventCreate(&kernel_event_stop);
    hipEventCreate(&it_start);
    hipEventCreate(&it_stop);

    while (!stop) {
        stop = it_count > max_steps;
        //region PointersReassign
        auto temp_h = d_old_H;
        auto temp_u = d_old_U;
        auto temp_w = d_old_W;
        auto temp_v = d_old_V;

        d_old_H = d_H;
        d_old_U = d_U;
        d_old_W = d_W;
        d_old_V = d_V;

        d_H = temp_h;
        d_U = temp_u;
        d_W = temp_w;
        d_V = temp_v;
        //endregion PointersReassign
        hipEventRecord(it_start);

        hipEventRecord(kernel_event_start);
        h_kernel<<<num_blocks, num_threads_per_block>>>(d_old_H, d_old_W, d_old_V, d_H, d_sim_params);
        hipEventRecord(kernel_event_stop);
        hipEventSynchronize(kernel_event_stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, kernel_event_start, kernel_event_stop);
        h_times.push_back(milliseconds);

        hipEventRecord(kernel_event_start);
        u_kernel<<<num_blocks, num_threads_per_block>>>(d_H, d_U, d_sim_params);
        hipEventRecord(kernel_event_stop);
        hipEventSynchronize(kernel_event_stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, kernel_event_start, kernel_event_stop);
        u_times.push_back(milliseconds);

        hipEventRecord(kernel_event_start);
        w_kernel<<<num_blocks, num_threads_per_block>>>(d_H, d_old_W, d_old_V, d_W, d_sim_params);
        hipEventRecord(kernel_event_stop);
        hipEventSynchronize(kernel_event_stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, kernel_event_start, kernel_event_stop);
        w_times.push_back(milliseconds);
        hipDeviceSynchronize();

        hipEventRecord(kernel_event_start);
        v_func_kernel<<<num_blocks, num_threads_per_block>>>(d_W, d_V, d_U, d_sim_params);
        integrate_v_kernel<<<num_blocks, num_threads_per_block>>>(d_V, d_sim_params);
        hipEventRecord(kernel_event_stop);
        hipEventSynchronize(kernel_event_stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, kernel_event_start, kernel_event_stop);
        v_times.push_back(milliseconds);

        hipEventRecord(it_stop);
        hipEventSynchronize(it_stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, it_start, it_stop);
        it_times.push_back(milliseconds);

//        std::cout << "-----------------------------------------------" << std::endl;
//        std::cout << " Starting iteration " << it_count++ << std::endl;
//        hipMemcpy(H, d_old_H, grid_size_bytes, hipMemcpyDeviceToHost);
//        hipMemcpy(U, d_old_U, grid_size_bytes, hipMemcpyDeviceToHost);
//        hipMemcpy(W, d_old_W, grid_size_bytes, hipMemcpyDeviceToHost);
//        hipMemcpy(V, d_old_V, grid_size_bytes, hipMemcpyDeviceToHost);
//        for(int i = 0; i < sim_params.dims[0] * sim_params.dims[1] * sim_params.dims[2]; i++){
//            auto id = indexof(i, &sim_params);
//            std::cout << "x: " << id.x << " y: " << id.y << " z: " << id.z << " v: " << H[i] << std::endl;
//        }
//        if (it_count == 3)
//            return
//        print_min_max_values(U, "u", sim_params);
//        print_min_max_values(V, "v", sim_params);
//        print_min_max_values(W, "w", sim_params);
//        print_min_max_values(H, "h", sim_params);

//        ss.str(std::string());
//        ss << filename << std::setfill('0') << std::setw(5) << it_count << ".vts";
//        export_vector_field(ss.str(), U, V, W, sim_params);
//        it_count++;

        if (it_count++ % print_every == 0) {
            std::cout << "-----------------------------------------------" << std::endl;
            std::cout << " Starting iteration " << it_count << std::endl;
            std::cout << "Average h_kernel: " << std::accumulate(h_times.begin(), h_times.end(), 0.0) / h_times.size()
                      << std::endl;
            std::cout << "Average u_kernel: " << std::accumulate(u_times.begin(), u_times.end(), 0.0) / u_times.size()
                      << std::endl;
            std::cout << "Average w_kernel: " << std::accumulate(w_times.begin(), w_times.end(), 0.0) / w_times.size()
                      << std::endl;
            std::cout << "Average v_kernel: " << std::accumulate(v_times.begin(), v_times.end(), 0.0) / v_times.size()
                      << std::endl;
            std::cout << "Average it time: " << std::accumulate(it_times.begin(), it_times.end(), 0.0) / it_times.size()
                      << std::endl;
        }

        if (it_count % save_every == 0) {
            hipMemcpy(H, d_old_H, grid_size_bytes, hipMemcpyDeviceToHost);
            hipMemcpy(U, d_old_U, grid_size_bytes, hipMemcpyDeviceToHost);
            hipMemcpy(W, d_old_W, grid_size_bytes, hipMemcpyDeviceToHost);
            hipMemcpy(V, d_old_V, grid_size_bytes, hipMemcpyDeviceToHost);

            print_min_max_values(U, "u", sim_params);
            print_min_max_values(V, "v", sim_params);
            print_min_max_values(W, "w", sim_params);
            print_min_max_values(H, "h", sim_params);

            ss.str(std::string());
            ss << filename << std::setfill('0') << std::setw(5) << it_count << ".vts";
            export_vector_field(ss.str(), U, V, W, sim_params);
        }

        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stdout, "Error occured, %s \n", hipGetErrorString(err));
        }

    }

    hipFree(d_H);
    hipFree(d_U);
    hipFree(d_W);
    hipFree(d_V);
    hipFree(d_old_H);
    hipFree(d_old_U);
    hipFree(d_old_W);
    hipFree(d_old_V);
    hipFree(d_sim_params);

    delete[] U;
    delete[] V;
    delete[] W;
    delete[] H;

}